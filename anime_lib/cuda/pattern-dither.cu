
#include <hip/hip_runtime.h>
// experimental pattern dithering cuda shader. it ain't great but it works?

extern "C" __device__ float clamp(float a, float min, float max) {
    if (a < min) {
        return min;
    } else if (a > max) {
        return max;
    } else {
        return a;
    }
}

// literally just bubblesort 
extern "C" __device__ void sort(float arr[64][4]) {
    for (int step = 0; step < 63; ++step) {
        bool swapped = false;

        for (int i= 0; i < 63 - step; ++i) {
            if (arr[i][3] > arr[i + 1][3]) {
                swapped = true;
                for (int j = 0; j < 3; ++j) {
                    float tmp = arr[i][j];
                    arr[i][j] = arr[i+1][j];
                    arr[i+1][j] = tmp;
                }
            }
        }

        if (!swapped) {
            break;
        }
    }
}

extern "C" __global__ void delta_e(const float* palette, const float* rgb_palette, const float* pixel, float* out, int H, int W, int matrix_size, int* matrix, float multiplier) {
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    int offset = (row * W + col) * 3;

    if ((row < H) && (col < W)) {
        float candidates[64][4];

        float acc[3] = {0.0,0.0,0.0};

        float src_r = pixel[offset];
        float src_g = pixel[offset + 1];
        float src_b = pixel[offset + 2];

        int bayer_idx = matrix[
            (row % 8)
            * 8
            + (col % 8)
        ];

        for (int j = 0; j < matrix_size; ++j) {
            float r = clamp(src_r + (acc[0] * multiplier), 0.0, 255.0) / 255.0;
            float g =  clamp(src_g + (acc[1] * multiplier), 0.0, 255.0) / 255.0;
            float b =  clamp(src_b + (acc[2] * multiplier), 0.0, 255.0) / 255.0;
            // printf("FIRST src_r %f src_g %f src_b %f \n r %f g %f b %f\n",src_r,src_g,src_b,r,g,b);

            if (r > 0.04045) {
                r = __powf(((r + 0.055) / 1.055), 2.4);
            } else {
                r = r / 12.92;
            }

            r = r * 100.0;

            if (g > 0.04045) {
                g = __powf(((g + 0.055) / 1.055), 2.4);
            } else {
                g = g / 12.92;
            }

            g = g * 100.0;

            if (b > 0.04045) {
                b = __powf(((b + 0.055) / 1.055), 2.4);
            } else {
                b = b / 12.92;
            }

            b = b * 100.0;

            // printf("src_r %f src_g %f src_b %f\n linearized: r %f g %f b %f\n",src_r,src_g,src_b,r,g,b);


            float x = r * 0.4124 + g * 0.3576 + b * 0.1805;
            float y = r * 0.2166 + g * 0.7152 + b * 0.0722;
            float z = r * 0.0193 + g * 0.1192 + b * 0.9505;

            x = x / 95.047;
            if (x > 0.008856) {
                x = __powf(x, 1.0 / 3.0);
            } else {
                x = (x * 7.787) + 16.0 / 116.0;
            }

            y = y / 100.000;

            if (y > 0.008856) {
                y = __powf(y, 1.0 / 3.0);
            } else {
                y = (y * 7.787) + 16.0 / 116.0;
            }

            z = z / 108.883;

            if (z > 0.008856) {
                z = __powf(z, 1.0 / 3.0);
            } else {
                z = (z * 7.787) + 16.0 / 116.0;
            }

            float l_l = (y * 116.0) - 16.0;
            float l_a = (x - y) * 500.0;
            float l_b = (y - z) * 200.0;

            float least_dist = 10000000000000000000000.0;
            int least_index = 276;

            for (int i = 0; i < 256; ++i) {
                int pal_offset = i * 3;
                float distl = palette[pal_offset] - l_l;
                float dista = palette[pal_offset + 1] - l_a;
                float distb = palette[pal_offset + 2] - l_b;
                float dist = (distl * distl) + (dista * dista) + (distb * distb);
                if (dist < least_dist) {
                    least_dist = dist;
                    least_index = i;
                }
            }
                        
            int pal_offset = least_index * 3;
            float p_r = rgb_palette[pal_offset];
            float p_g = rgb_palette[pal_offset + 1];
            float p_b = rgb_palette[pal_offset + 2];

            candidates[j][0] = p_r;
            candidates[j][1] = p_g;
            candidates[j][2] = p_b;

            candidates[j][3] = (p_r * 299.0 + p_g * 587.0 + p_b * 114.0) / (255.0 * 1000.0);

            acc[0] = acc[0] + (src_r - p_r);
            acc[1] = acc[1] + (src_g - p_g);
            acc[2] = acc[2] + (src_b - p_b);
        }

        sort(candidates);

        out[offset] = candidates[bayer_idx][0];
        out[offset + 1] = candidates[bayer_idx][1];
        out[offset + 2] = candidates[bayer_idx][2];
    }
}